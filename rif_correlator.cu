#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdlib.h>
#include <stdio.h>

#include <sys/socket.h>
#include <netinet/in.h>
#include <stdio.h>
#include <stdlib.h>
#include <strings.h>

int imin(int a, int b)
{
	return (a < b ? a : b);
}

/* UDP port */
#define UDP_PORT_NUMBER 32000

/* Number of samples per spectrum */
#define NX 1024

/* Number of samples to average per channel per time bin */
#define N 19999744
/*20e6*/

/* Number of FFTs to perform in one batch */
#define BATCH (N / NX)

/* Number of samples after FFT */
#define NF (NX/2+1)*BATCH

/* Dimensions for thread blocks */
const int threadsPerBlock = 1024;
const int minBlocksPerGrid = 512;
const int blocksPerGrid = imin(minBlocksPerGrid, (NF+threadsPerBlock-1) / threadsPerBlock);

__global__ void correlate(float *c, float *s, hipfftComplex *a, hipfftComplex *b)
{
	__shared__ float cache_c[threadsPerBlock];
	__shared__ float cache_s[threadsPerBlock];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	float temp_c = 0;
	float temp_s = 0;

	hipfftComplex corr;

	while (tid < NF) {
		/* Normalize FFT */
		a[tid].x /= NX;
		a[tid].y /= NX;
		b[tid].x /= NX;
		b[tid].y /= NX;

		corr = hipCmulf(a[tid], hipConjf(b[tid]));
		
		temp_c += hipCrealf(corr);
		temp_s += hipCimagf(corr);

		tid += blockDim.x * gridDim.x;
	}

	cache_c[cacheIndex] = temp_c;
	cache_s[cacheIndex] = temp_s;

	__syncthreads();

	/* Average values in cache */
	int i = blockDim.x / 2;
	while (i != 0) {
		if (cacheIndex < i) {
			cache_c[cacheIndex] += cache_c[cacheIndex + i];
			cache_s[cacheIndex] += cache_s[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	/* Store the result */
	if (cacheIndex == 0) {
		c[blockIdx.x] = cache_c[0];
		s[blockIdx.x] = cache_s[0];
	}
}

int main(int argc, char* argv[])
{
	int i, j;
	FILE *fo;
	char *buffer;
	float c, s;
	float *a, *b, *partial_c, *partial_s, *dev_a, *dev_b, *dev_partial_c, *dev_partial_s;
	hipfftComplex *cdev_a, *cdev_b;
	hipError_t err;
	hipfftHandle plan;

  int sockfd, n;
  struct sockaddr_in servaddr, cliaddr;
  char mesg[1000];

  if (argc != 2)
  {
    printf("usage: rif_correlator <file>\n");
    exit(1);
  }

	printf("threadsPerBlock: %d\n", threadsPerBlock);
	printf("minBlocksPerGrid: %d\n", minBlocksPerGrid);
	printf("blocksPerGrid: %d\n", blocksPerGrid);

  /* Setup UDP port */
  sockfd=socket(AF_INET, SOCK_DGRAM, 0);

  bzero(&servaddr, sizeof(servaddr));
  servaddr.sin_family = AF_INET;
  servaddr.sin_addr.s_addr=htonl(INADDR_ANY);
  servaddr.sin_port=htons(UDP_PORT_NUMBER);
  bind(sockfd, (struct sockaddr *)&servaddr, sizeof(servaddr));

	/* Allocate memory on host */
	buffer = (char*) malloc(2*N*sizeof(char));
	a = (float*) malloc(N*sizeof(float));
	b = (float*) malloc(N*sizeof(float));
	partial_c = (float*) malloc(blocksPerGrid*sizeof(float));
	partial_s = (float*) malloc(blocksPerGrid*sizeof(float));

	/* Allocate memory on device */
	err = hipMalloc(&dev_partial_c, blocksPerGrid*sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "Error %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc(&dev_partial_s, blocksPerGrid*sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "Error %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc(&dev_a, N*sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "Error %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc(&dev_b, N*sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "Error %s\n", hipGetErrorString(err));
		return 1;
	}
	
	err = hipMalloc(&cdev_a, (NX/2+1)*BATCH*sizeof(hipfftComplex));
	if (err != hipSuccess) {
		fprintf(stderr, "Error %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMalloc(&cdev_b, (NX/2+1)*BATCH*sizeof(hipfftComplex));
	if (err != hipSuccess) {
		fprintf(stderr, "Error %s\n", hipGetErrorString(err));
		return 1;
	}

	/* Create FFT plan */
	if (hipfftPlan1d(&plan, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return 1;	
	}

	/* Open output file */
	fo = fopen(argv[1], "w");
	if (fo == NULL) {
		fprintf(stderr, "Error: could not open output file!\n");
		return 1;
	}

	i = 0;
	for (;;) {
    n = recvfrom(sockfd, buffer, 2*N*sizeof(char), 0, (struct sockaddr *)&cliaddr, sizeof(cliaddr));

		printf("%d\n", i);

		/* Copy data to device */
		for (j=0; j<N; j++) {
			a[j] = (float) buffer[2*j];
			b[j] = (float) buffer[2*j+1];
		}

		err = hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			printf("Error %s\n", hipGetErrorString(err));
		}
	
		err = hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			printf("Error %s\n", hipGetErrorString(err));
		}

		/* Perform FFT on device */
		if (hipfftExecR2C(plan, dev_a, cdev_a) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
			return 1;	
		}

		if (hipDeviceSynchronize() != hipSuccess){
			fprintf(stderr, "Cuda error: Failed to synchronize\n");
			return 1;	
		}

		/* Perform FFT on device */
		if (hipfftExecR2C(plan, dev_b, cdev_b) != HIPFFT_SUCCESS){
			fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
			return 1;	
		}

		if (hipDeviceSynchronize() != hipSuccess){
			fprintf(stderr, "Cuda error: Failed to synchronize\n");
			return 1;	
		}

		correlate<<<blocksPerGrid,threadsPerBlock>>>(dev_partial_c, dev_partial_s, cdev_a, cdev_b);
	
		err = hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Error %s\n", hipGetErrorString(err));
		}

		err = hipMemcpy(partial_s, dev_partial_s, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			printf("Error %s\n", hipGetErrorString(err));
		}

		/* Finish partial sums on the CPU */
		c = 0;
		s = 0;
		for (j=0; j<blocksPerGrid; j++) {
			c += partial_c[j];
			s += partial_s[j];
		}
		c /= BATCH;
		s /= BATCH;

		fprintf(fo, "%.3f\t%.3f\n", c, s);

		i++;
	}

	/* Cleanup */
	free(a);
	free(b);
	free(buffer);
	hipfftDestroy(plan);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(cdev_a);
	hipFree(cdev_b);

	/* Close file */
	fclose(fo);
	return 0;
}

